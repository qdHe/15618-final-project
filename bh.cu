#include "hip/hip_runtime.h"
/*
0. Read input data and transfer to GPU
for each timestep do {
1. Compute bounding box around all bodies
2. Build hierarchical decomposition by inserting each body into octree
3. Summarize body information in each internal octree node
4. Approximately sort the bodies by spatial distance
5. Compute forces acting on each body with help of octree
6. Update body positions and velocities
}
7. Transfer result to CPU and output
*/
#include <iostream>
#include <fstream>
//#include <chrono>
#include <algorithm>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>


#define TOTAL_V_NUM 10000
#define CELL_NUM 4
#define LOCK -2
#define NOTHING -1

#ifdef DEBUG
#define DEBUG_PRINT(fmt, args...)    printf(fmt, ## args)
#else
# define DEBUG_PRINT(fmt, args...) do {} while (false)
#endif

float find_min(float* nums, int N){
    thrust::device_ptr<float> ptr(nums);
    int result_offset = thrust::min_element(ptr, ptr + N) - ptr;
    float min_x = *(ptr + result_offset);
    printf("min: %f\n", min_x);
    return min_x;
}

float find_max(float* nums, int N){
    thrust::device_ptr<float> ptr(nums);
    int result_offset = thrust::max_element(ptr, ptr + N) - ptr;
    float max_x = *(ptr + result_offset);
    return max_x;
}

void batch_set(int* raw_ptr, int N, int target){
    thrust::device_ptr<int> dev_ptr(raw_ptr);
    thrust::fill(dev_ptr, dev_ptr + N, (int) target);
}

__global__ void BuildTreeKernel(float* posx, float* posy, int* child, int* _bottom, float radius, int N, int node_num,
                     float rootx, float rooty){
    int threadId = blockIdx.x*blockDim.x + threadIdx.x;
    int threadNum = blockDim.x*gridDim.x;//TODO
    int step = threadNum;
    bool newBody = true;
    float x;
    float y;
    float curRad;
    int rootIndex;
    int curIndex;
    int lastIndex;
    int i = threadId;
    int path;
    int locked;
    int old_node_path;
    rootIndex = node_num;
    if(threadId == 0){
        posx[rootIndex] = rootx;
        posy[rootIndex] = rooty;
        printf("bottom %d\n",*_bottom);
    }

    while(i < N){
        // initialize

        if(newBody){
            x = posx[i];
            y = posy[i];
            path = 0;
            if(rootx < x){
                path += 1;
            }
            if(rooty < y){
                path += 2;
            }
            lastIndex = rootIndex;
            newBody = false;
            curRad = radius;
            printf("new body %d x %f y %f\n", i, x, y);
        }
        curIndex = child[CELL_NUM*lastIndex+path];//TODO
        while(curIndex >= N){
            lastIndex = curIndex;
            path = 0;
            if(posx[lastIndex] < x){
                path += 1;
            }
            if(posx[lastIndex] < y){
                path += 2;
            }
            curIndex = child[CELL_NUM*curIndex+path];
            curRad *= 0.5;
        }

        if (curIndex != LOCK) {
            locked = CELL_NUM*lastIndex+path;
            if (curIndex == atomicCAS((int*)&child[locked], curIndex, LOCK)) {
                printf("add to tree node %d posx %f posy %f\n", lastIndex, posx[lastIndex], posy[lastIndex]);
                if (curIndex == NOTHING) {
                    child[locked] = i; // insert body and release lock
                    printf("empty, add %d to %d path %d success\n", i, lastIndex, path);
                } else {
                    int old_node = curIndex;
                    float old_node_x = posx[old_node];
                    float old_node_y = posy[old_node];
                    int cell = atomicSub((int*)_bottom, 1) - 1;
                    int new_cell = cell;
                    printf("old node %d x:%f y:%f new cell:%d\n", old_node, old_node_x, old_node_y, cell);
                    do{
                        if(cell<N){
                            printf("error break\n");
                            break;
                        }
                        child[CELL_NUM*lastIndex+path] = cell;
                        posx[cell] = posx[lastIndex] - curRad*0.5 + (path&1)*curRad;
                        posy[cell] = posy[lastIndex] - curRad*0.5 + ((path>>1)&1)*curRad;

                        curRad *= 0.5;

                        path = 0;
                        if(posx[cell] < x){
                            path += 1;
                        }
                        if(posx[cell] < y){
                            path += 2;
                        }
                        old_node_path = 0;
                        if(posx[cell] < old_node_x){
                            old_node_path += 1;
                        }
                        if(posx[cell] < old_node_y){
                            old_node_path += 2;
                        }
                        printf("new cell %d x:%f y:%f rad:%f old id %d path %d new id %d path %d\n",
                                cell, posx[cell], posy[cell], curRad, old_node, old_node_path, i, path);
                        if(path != old_node_path){
                            child[cell*CELL_NUM+path] = i;
                            child[cell*CELL_NUM+old_node_path] = old_node;
                            printf("new cell %d x:%f y:%f rad:%f old id %d path %d new id %d path %d break\n",
                                   cell, posx[cell], posy[cell], curRad, old_node, old_node_path, i, path);
                            break;
                        }else{
                            lastIndex = cell;
                            cell = atomicSub((unsigned int*)_bottom, 1) - 1;
                        }
                    }while(true);
                    __threadfence();
                    child[locked] = new_cell;
                }
                newBody = true;
                i += step;
            }
        }
        //__syncthreads();
    }


}

__global__ void SummarizeTreeKernel(float* posx, float* posy, int* child, int* count, int* _bottom, int node_num, int N){
    
    int missing = 0;
    int child_node;
    int cache[CELL_NUM] = {0};
    int cache_tail = 0;
    int step = blockDim.x*gridDim.x;//TODO

    int child_num;
    int tmp_count;
    int tmp_c;
    float sum_x;
    float sum_y;
    int threadId = blockIdx.x*blockDim.x + threadIdx.x;
    int node_id = threadId + *_bottom;
    if(threadId == 0){
        for(int i=0; i<N; i++){
            printf("count %d:%d\n", i, count[i]);
        }
    }

    while(node_id<node_num){
        
        if(missing == 0){
            child_num = 0;
            sum_x = 0.0;
            sum_y = 0.0;
            tmp_count = 0;
            cache_tail = 0;
            for(int i=CELL_NUM*node_id; i<CELL_NUM*node_id+CELL_NUM; i++){
                int child_node = child[i];
                if(child_node >= 0){
                    if(count[child_node] > 0){
                        sum_x += posx[child_node];
                        sum_y += posy[child_node];
                        tmp_count += count[child_node];
                    }else{
                        missing++;
                        //TODO cache index
                        cache[cache_tail++] = child_node;
                    }
                    child_num++;
                }
                
            }
        }
        if(missing != 0){

            do{
                child_node = cache[cache_tail-1];
                tmp_c = count[child_node];
                if(tmp_c > 0){
                    missing--;
                    sum_x += posx[child_node];
                    sum_y += posy[child_node];
                    tmp_count += tmp_c;
                    cache_tail--;
                }
            }while(missing != 0 && tmp_c > 0);
        }
        if(missing == 0){
            printf("%d before: x %f y %f\n", node_id, posx[node_id], posy[node_id]);
            posx[node_id] = sum_x/tmp_count;
            posy[node_id] = sum_y/tmp_count;
            //FENCE
            __threadfence();
            count[node_id] = tmp_count;
            printf("%d after: x %f y %f count %d\n", node_id, posx[node_id], posy[node_id], count[node_id]);
            node_id += step;
        }
    }
}

void printTree(float* posx, float* posy, int* child, int node_num, int N){
    bool flag;
    for(int i=node_num; i>=N; i--){
        flag = false;
        printf("node %d posx %f posy %f \n", i, posx[i], posy[i]);
        for(int j=i*CELL_NUM; j<i*CELL_NUM+CELL_NUM; j++){

            printf(" child:%d", child[j]);
            if(child[j] > 0) flag=true;
        }

        printf("\n");
        if(!flag) break;
    }
    for(int i=0; i<N; i++){
        printf("node %d posx %f posy %f \n", i, posx[i], posy[i]);
    }
}

void BH(float* hostx, float* hosty, int N, int timesteps){
    int gridDim = 16;
    int blockDim = 16;
    float* posx;
    float* posy;
    int* child;
    int* count;
    int* _bottom;
    float minx, miny;
    float maxx, maxy;
    float radius;
    float rootx;
    float rooty;
    int node_num = N*2;
    int iter;
    //===debug===//
    int* host_child = new int [(node_num+1)*CELL_NUM];

    hipMalloc(&posx, sizeof(float)*(node_num+1));
    hipMalloc(&posy, sizeof(float)*(node_num+1));
    hipMalloc(&child, sizeof(int)*CELL_NUM*(node_num+1));


    hipMalloc(&count, sizeof(int)*(node_num+1));
    hipMalloc(&_bottom, sizeof(int));
    //INIT
    hipMemcpy(posx, hostx, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(posy, hosty, sizeof(float)*N, hipMemcpyHostToDevice);

    //FORCE DIRECTED
    for (iter = 0; iter < timesteps; iter++) {
        //Calculating repulsive force
        //Calculating bounding box
        printf("iter %d\n", iter);
        minx = find_min(posx, N);
        maxx = find_max(posx, N);
        miny = find_min(posy, N);
        maxy = find_max(posy, N);
        radius = std::max(maxx-minx,maxy-miny);
        radius *= 0.5;
        rootx = (minx+maxx)*0.5;
        rooty = (miny+maxy)*0.5;
        printf("rootx %f rooty %f radius %f\n", rootx, rooty, radius);
        hipMemset(_bottom, node_num, 1);
        //Build Tree
        batch_set(child, CELL_NUM*(node_num+1), -1);
        BuildTreeKernel<<<gridDim, blockDim>>>(posx, posy, child, _bottom, radius, N, node_num, rootx, rooty);
        hipDeviceSynchronize();
        printf("build tree success!\n");

        //===debug====//
        hipMemcpy(host_child, child, sizeof(int)*CELL_NUM*(node_num+1),hipMemcpyDeviceToHost);
        hipMemcpy(hostx, posx, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        hipMemcpy(hosty, posy, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        printTree(hostx, hosty, host_child, node_num, N);


        //Summerize Tree
        batch_set(count+N, node_num-N, -1);
        batch_set(count, N, 1);
        SummarizeTreeKernel<<<gridDim, blockDim>>>(posx, posy, child, count, _bottom, node_num, N);
        hipDeviceSynchronize();

        //===debug====//
        hipMemcpy(host_child, child, sizeof(int)*CELL_NUM*(node_num+1),hipMemcpyDeviceToHost);
        hipMemcpy(hostx, posx, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        hipMemcpy(hosty, posy, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        printTree(hostx, hosty, host_child, node_num, N);
        //Compute force

        //Calculating attractive force

        //Update


    }
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(){
    int N = 8;
    float *hostx = new float[N];
    float *hosty = new float[N];
    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
    for(int i=0; i<N; i++){
        hostx[i] = i*0.1f;//(float(rand())/RAND_MAX-0.5f);
        hosty[i] = i*0.1f;(float(rand())/RAND_MAX-0.5f);
    }
    BH(hostx, hosty, N, 1);
}

