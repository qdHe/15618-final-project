#include "hip/hip_runtime.h"
/*
0. Read input data and transfer to GPU
for each timestep do {
1. Compute bounding box around all bodies
2. Build hierarchical decomposition by inserting each body into octree
3. Summarize body information in each internal octree node
4. Approximately sort the bodies by spatial distance
5. Compute forces acting on each body with help of octree
6. Update body positions and velocities
}
7. Transfer result to CPU and output
*/
#include <iostream>
#include <fstream>
//#include <chrono>
#include <algorithm>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>


#define TOTAL_V_NUM 10000
#define CELL_NUM 4
#define LOCK -2
#define NOTHING -1
#define MAXDEPTH 10
#define THREADS5 16
#define WARPSIZE 32
#define W 10
#define H 10
int N = 8;//tentative
int M = 8;
float K = sqrt(1.0*W*H/N);

#ifdef DEBUG
#define DEBUG_PRINT(fmt, args...)    printf(fmt, ## args)
#else
# define DEBUG_PRINT(fmt, args...) do {} while (false)
#endif

struct GlobalConstants{
    int N;
    int M;
    float K;
    float Alpha;
    float Eps;
    float Thr;
};

float find_min(float* nums, int N){
    thrust::device_ptr<float> ptr(nums);
    int result_offset = thrust::min_element(ptr, ptr + N) - ptr;
    float min_x = *(ptr + result_offset);
    printf("min: %f\n", min_x);
    return min_x;
}

float find_max(float* nums, int N){
    thrust::device_ptr<float> ptr(nums);
    int result_offset = thrust::max_element(ptr, ptr + N) - ptr;
    float max_x = *(ptr + result_offset);
    return max_x;
}

void batch_set(int* raw_ptr, int N, int target){
    thrust::device_ptr<int> dev_ptr(raw_ptr);
    thrust::fill(dev_ptr, dev_ptr + N, (int) target);
}
__constant__ GlobalConstants deviceParams;
__global__ void BuildTreeKernel(float* posx, float* posy, int* child, int* _bottom, float radius, int N, int node_num,
                                float rootx, float rooty){
    int threadId = blockIdx.x*blockDim.x + threadIdx.x;
    int threadNum = blockDim.x*gridDim.x;//TODO
    int step = threadNum;
    bool newBody = true;
    float x;
    float y;
    float curRad;
    int rootIndex;
    int curIndex;
    int lastIndex;
    int i = threadId;
    int path;
    int locked;
    int old_node_path;
    rootIndex = node_num;
    if(threadId == 0){
        posx[rootIndex] = rootx;
        posy[rootIndex] = rooty;
        printf("bottom %d\n",*_bottom);
    }

    while(i < N){
        // initialize

        if(newBody){
            x = posx[i];
            y = posy[i];
            path = 0;
            if(rootx < x){
                path += 1;
            }
            if(rooty < y){
                path += 2;
            }
            lastIndex = rootIndex;
            newBody = false;
            curRad = radius;
            printf("new body %d x %f y %f\n", i, x, y);
        }
        curIndex = child[CELL_NUM*lastIndex+path];//TODO
        while(curIndex >= N){
            lastIndex = curIndex;
            path = 0;
            if(posx[lastIndex] < x){
                path += 1;
            }
            if(posx[lastIndex] < y){
                path += 2;
            }
            curIndex = child[CELL_NUM*curIndex+path];
            curRad *= 0.5;
        }

        if (curIndex != LOCK) {
            locked = CELL_NUM*lastIndex+path;
            if (curIndex == atomicCAS((int*)&child[locked], curIndex, LOCK)) {
                printf("add to tree node %d posx %f posy %f\n", lastIndex, posx[lastIndex], posy[lastIndex]);
                if (curIndex == NOTHING) {
                    child[locked] = i; // insert body and release lock
                    printf("empty, add %d to %d path %d success\n", i, lastIndex, path);
                } else {
                    int old_node = curIndex;
                    float old_node_x = posx[old_node];
                    float old_node_y = posy[old_node];
                    int cell = atomicSub((int*)_bottom, 1) - 1;
                    int new_cell = cell;
                    printf("old node %d x:%f y:%f new cell:%d\n", old_node, old_node_x, old_node_y, cell);
                    do{
                        if(cell<N){
                            printf("error break\n");
                            break;
                        }
                        child[CELL_NUM*lastIndex+path] = cell;
                        posx[cell] = posx[lastIndex] - curRad*0.5 + (path&1)*curRad;
                        posy[cell] = posy[lastIndex] - curRad*0.5 + ((path>>1)&1)*curRad;

                        curRad *= 0.5;

                        path = 0;
                        if(posx[cell] < x){
                            path += 1;
                        }
                        if(posx[cell] < y){
                            path += 2;
                        }
                        old_node_path = 0;
                        if(posx[cell] < old_node_x){
                            old_node_path += 1;
                        }
                        if(posx[cell] < old_node_y){
                            old_node_path += 2;
                        }
                        printf("new cell %d x:%f y:%f rad:%f old id %d path %d new id %d path %d\n",
                               cell, posx[cell], posy[cell], curRad, old_node, old_node_path, i, path);
                        if(path != old_node_path){
                            child[cell*CELL_NUM+path] = i;
                            child[cell*CELL_NUM+old_node_path] = old_node;
                            printf("new cell %d x:%f y:%f rad:%f old id %d path %d new id %d path %d break\n",
                                   cell, posx[cell], posy[cell], curRad, old_node, old_node_path, i, path);
                            break;
                        }else{
                            lastIndex = cell;
                            cell = atomicSub((unsigned int*)_bottom, 1) - 1;
                        }
                    }while(true);
                    __threadfence();
                    child[locked] = new_cell;
                }
                newBody = true;
                i += step;
            }
        }
        //__syncthreads();
    }


}

__global__ void SummarizeTreeKernel(float* posx, float* posy, int* child, int* count, int* _bottom, int node_num){

    int missing = 0;
    int child_node;
    int cache[CELL_NUM] = {0};
    int cache_tail = 0;
    int step = blockDim.x*gridDim.x;//TODO

    int child_num;
    int tmp_count;
    int tmp_c;
    float sum_x;
    float sum_y;
    int threadId = blockIdx.x*blockDim.x + threadIdx.x;
    int node_id = threadId + *_bottom;

    while(node_id<node_num){

        if(missing == 0){
            child_num = 0;
            sum_x = 0.0;
            sum_y = 0.0;
            tmp_count = 0;
            cache_tail = 0;
            for(int i=CELL_NUM*node_id; i<CELL_NUM*node_id+CELL_NUM; i++){
                int child_node = child[i];
                if(child_node >= 0){
                    if(count[child_node] > 0){
                        sum_x += posx[child_node];
                        sum_y += posy[child_node];
                        tmp_count += count[child_node];
                    }else{
                        missing++;
                        //TODO cache index
                        cache[cache_tail++] = child_node;
                    }
                    child_num++;
                }

            }
        }
        if(missing != 0){

            do{
                child_node = cache[cache_tail-1];
                tmp_c = count[child_node];
                if(tmp_c > 0){
                    missing--;
                    sum_x += posx[child_node];
                    sum_y += posy[child_node];
                    tmp_count += tmp_c;
                    cache_tail--;
                }
            }while(missing != 0 && tmp_c > 0);
        }
        if(missing == 0){
            printf("%d before: x %f y %f\n", node_id, posx[node_id], posy[node_id]);
            posx[node_id] = sum_x/tmp_count;
            posy[node_id] = sum_y/tmp_count;
            //FENCE
            __threadfence();
            count[node_id] = tmp_count;
            printf("%d after: x %f y %f count\n", node_id, posx[node_id], posy[node_id], count[node_id]);
            node_id += step;
        }
    }
}

void printTree(float* posx, float* posy, int* child, int node_num, int N){
    bool flag;
    for(int i=node_num; i>=N; i--){
        flag = false;
        printf("node %d posx %f posy %f \n", i, posx[i], posy[i]);
        for(int j=i*CELL_NUM; j<i*CELL_NUM+CELL_NUM; j++){

            printf(" child:%d", child[j]);
            if(child[j] > 0) flag=true;
        }

        printf("\n");
        if(!flag) break;
    }
    for(int i=0; i<N; i++){
        printf("node %d posx %f posy %f \n", i, posx[i], posy[i]);
    }
}

/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/

__global__ void SortKernel(int* startd, int *sort, int *child, int *count, 
			int *_bottom, int node_num) {

	int N = deviceParams.N;
	int bottom = *_bottom;
    int gridSize = blockDim.x * gridDim.x;
    int cell = node_num + 1 - gridSize + threadIdx.x + blockIdx.x * blockDim.x;

    // iterate over all cells assigned to thread
    while (cell >= bottom) {
        int start = startd[cell];
        if (start >= 0) {
            for (int i = 0; i < 4; ++i) {
                int childIdx = child[cell*4+i];
				if(childIdx == 15){
					printf("child[15], count = %d\n", count[15]);
				}
                if (childIdx >= N) {
					printf("   #case1: start = %d, i = %d, childIdx = %d\n",
								start, i, childIdx);
                    // child is a cell
                    startd[childIdx] = start;  // set start ID of child
                    start += count[childIdx];  // add #bodies in subtree
                } else if (childIdx >= 0) {
					printf("   #case2: start = %d, i = %d, childIdx = %d\n",
								start, i, childIdx);
                    // child is a body
                    sort[start] = childIdx;  // record body in 'sorted' array
                    ++start;
                }
            }
            cell -= gridSize;  // move on to next cell
        }
        __syncthreads();  // throttle
    }
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/

__device__ __inline__ float repulsive_force(float dist){
    return deviceParams.K*deviceParams.K/dist/deviceParams.N/10000;
}

__device__ __inline__ float attractive_force(float dist){
    return dist*dist/deviceParams.K/deviceParams.N;
}

__global__
void ForceCalculationKernel(float* posx, float* posy, int* child, int* count, int *sort, int *E, int *Idx,
        float *dispX, float *dispY, int node_num, int *_maxdepth, float radius)
{
    __shared__ volatile int pos[MAXDEPTH * THREADS5/WARPSIZE], node[MAXDEPTH * THREADS5/WARPSIZE];
    __shared__ float dq[MAXDEPTH * THREADS5/WARPSIZE];
    float alpha = deviceParams.Alpha, eps = deviceParams.Eps;
	int maxdepth = *_maxdepth;

    if (0 == threadIdx.x) {
        // precompute values that depend only on tree level
        dq[0] = radius * radius * alpha;
        for (int i = 1; i < maxdepth; i++) {
            dq[i] = dq[i - 1] * 0.25f;
            dq[i - 1] += eps;
        }
        dq[maxdepth - 1] += eps;

        if (maxdepth > MAXDEPTH) {
            // error
            printf("ERROR");
        }
    }
    __syncthreads();

    if (maxdepth <= MAXDEPTH) {
        // figure out first thread in each warp (lane 0)
        int base = threadIdx.x / WARPSIZE;
        int sbase = base * WARPSIZE;
        int j = base * MAXDEPTH;

        int diff = threadIdx.x - sbase;
        // make multiple copies to avoid index calculations later
        if (diff < MAXDEPTH) {
            dq[diff+j] = dq[diff];
        }
        __syncthreads();

        // iterate over all bodies assigned to thread
        for (int k = threadIdx.x + blockIdx.x * blockDim.x; k < deviceParams.N; k += blockDim.x * gridDim.x) {
            int v = sort[k];  // get permuted/sorted index
            // cache position info
            float px = posx[v];
            float py = posy[v];
            float dispx = 0.;
            float dispy = 0.;
            // initialize iteration stack, i.e., push root node onto stack
            int depth = j;
            if (sbase == threadIdx.x) {
                node[j] = node_num;//nnodesd;
                pos[j] = 0;
            }

            while (depth >= j) {
                // stack is not empty
                int t;
                while ((t = pos[depth]) < 4) {
                    // node on top of stack has more children to process
                    int childIdx = child[node[depth]*4+t];  // load child pointer
                    if (sbase == threadIdx.x) {
                        // I'm the first thread in the warp
                        pos[depth] = t + 1;
                    }
                    if (childIdx >= 0) {
                        float dx = posx[childIdx] - px;
                        float dy = posy[childIdx] - py;
                        float dist = dx*dx + dy*dy + eps;  // compute distance squared (plus softening)
                        if ((childIdx < deviceParams.N) || __all(dist >= dq[depth])) {  // check if all threads agree that cell is far enough away (or is a body)
                            dist = rsqrtf(dist);  // compute distance
                            float rf = repulsive_force(dist)*count[childIdx];
                            dispx += dx/dist*rf;//disp_x
                            dispy += dy/dist*rf;//disp_y
                        } else {
                            // push cell onto stack
                            depth++;
                            if (sbase == threadIdx.x) {
                                node[depth] = childIdx;
                                pos[depth] = 0;
                            }
                        }
                    } else {
                        depth = max(j, depth - 1);  // early out because all remaining children are also zero
                    }
                }
                depth--;  // done with this level
            }

            int start = 0;
            if (v > 0){
                start = Idx[v-1];
            }
            for(int e=start; e<Idx[v]; e+=2){
                int u = E[e+1];
                float dx = px-posx[u];
                float dy = py-posy[u];
                float dist = sqrt(dx*dx+dy*dy+eps);
                float af = attractive_force(dist);
                //if(v==10) printf("u=%d, att_force= %f\n", u, af);
                dispx -= dx/dist*af;
                dispy -= dy/dist*af;
            }

            dispX[v] = dispx;
            dispY[v] = dispy;
        }
    }
}

__global__
void UpdatePosKernel(float* posx, float* posy, float *dispX, float *dispY){
    float  thr = deviceParams.Thr;
    float dispx, dispy, px, py;
    for (int v = threadIdx.x + blockIdx.x * blockDim.x; v < deviceParams.N; v += blockDim.x * gridDim.x) {
        dispx = dispX[v];
        dispy = dispY[v];
		px = posx[v];
		py = posy[v];
        float dist = sqrt(dispx * dispx + dispy * dispy);
        px += (dist > thr) ? dispx / dist * thr : dispx;
        py += (dist > thr) ? dispy / dist * thr : dispy;
        posx[v] = min(W / 2., max(-W / 2., px));
        posy[v] = min(H / 2., max(-H / 2., py));
    }
}


void BH(float* hostx, float* hosty, int *E, int *Idx, int N, int timesteps){
    int gridDim = 16;
    int blockDim = 16;
    float alpha = 4;
    float eps = 0.0025;
    float thr = W+H;
    float* posx;
    float* posy;
    float* dispx;
    float* dispy;
    int* child;
    int* count;
    int* debug_start = new int[2*N+1]();
    int* start;
    int* debug_sort = new int[N]();
    int* sort;
    int *deviceEdge;
    int *deviceIdx;
    int* _bottom;
    int *_maxdepth = new int(4); //TODO
    float minx, miny;
    float maxx, maxy;
    float radius;
    float rootx;
    float rooty;
    int node_num = N*2;
    int iter;

    //===debug===//
    int* host_child = new int [(node_num+1)*CELL_NUM];

    hipMalloc(&posx, sizeof(float)*(node_num+1));
    hipMalloc(&posy, sizeof(float)*(node_num+1));
    hipMalloc(&child, sizeof(int)*CELL_NUM*(node_num+1));
    hipMalloc(&start, sizeof(int)*(node_num+1));
    hipMalloc(&sort, sizeof(int)*N);
    hipMalloc(&deviceEdge, sizeof(int)*4*M);
    hipMalloc(&deviceIdx, sizeof(int)*N);
    hipMalloc(&dispx, sizeof(float)*N);
    hipMalloc(&dispy, sizeof(float)*N);

    hipMalloc(&count, sizeof(int)*(node_num+1));
    hipMalloc(&_bottom, sizeof(int));
    //INIT
    hipMemcpy(posx, hostx, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(posy, hosty, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(deviceEdge, E, sizeof(int)*4*M, hipMemcpyHostToDevice);
    hipMemcpy(deviceIdx, Idx, sizeof(int)*N, hipMemcpyHostToDevice);

    //FORCE DIRECTED
    for (iter = 0; iter < timesteps; iter++) {
        //Calculating repulsive force
        //Calculating bounding box
        printf("iter %d\n", iter);
        minx = find_min(posx, N);
        maxx = find_max(posx, N);
        miny = find_min(posy, N);
        maxy = find_max(posy, N);
        radius = std::max(maxx-minx,maxy-miny);
        radius *= 0.5;
        rootx = (minx+maxx)*0.5;
        rooty = (miny+maxy)*0.5;
        printf("rootx %f rooty %f radius %f\n", rootx, rooty, radius);
        hipMemset(_bottom, node_num, 1);

        GlobalConstants params;
        params.N = N;
        params.M = M;
        params.K = K;
        params.Thr = thr;
        params.Alpha = alpha;
        params.Eps = eps;
        hipMemcpyToSymbol(HIP_SYMBOL(deviceParams), &params, sizeof(GlobalConstants));



        //Build Tree
        batch_set(child, CELL_NUM*(node_num+1), -1);
        BuildTreeKernel<<<gridDim, blockDim>>>(posx, posy, child, _bottom, radius, N, node_num, rootx, rooty);
        hipDeviceSynchronize();
        printf("build tree success!\n");

        //===debug====//
        //hipMemcpy(host_child, child, sizeof(int)*CELL_NUM*(node_num+1),hipMemcpyDeviceToHost);
        //hipMemcpy(hostx, posx, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        //hipMemcpy(hosty, posy, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        //printTree(hostx, hosty, host_child, node_num, N);


        //Summerize Tree
        batch_set(count+N, node_num-N, -1);
        batch_set(count, N, 1);
        SummarizeTreeKernel<<<gridDim, blockDim>>>(posx, posy, child, count, _bottom, node_num);
        hipDeviceSynchronize();

        //===debug====//
        //hipMemcpy(host_child, child, sizeof(int)*CELL_NUM*(node_num+1),hipMemcpyDeviceToHost);
        //hipMemcpy(hostx, posx, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        //hipMemcpy(hosty, posy, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
        //printTree(hostx, hosty, host_child, node_num, N);
        //Sort
		hipMemset(start,0,sizeof(int)*(node_num+1));
        batch_set(start+N, N, -1);
        hipMemcpy(debug_start, start, sizeof(float)*(node_num+1), hipMemcpyDeviceToHost);
		for(int i = 0; i<=node_num; ++i){
			printf("%d ", debug_start[i]);
		}
        printf("\nPOS1\n");
		//printf("node_num: %d, bottom: %d\n", node_num, (*_bottom));
        SortKernel<<<gridDim, blockDim>>>(start, sort, child, count, _bottom, node_num);
        hipDeviceSynchronize();
        hipMemcpy(debug_sort, sort, sizeof(float)*N, hipMemcpyDeviceToHost);
		for(int i = 0; i<N; ++i){
			printf("%d ", debug_sort[i]);
		}
        printf("\nPOS2\n");
        //Compute force //TODO try separate repulsive force and attractive force calculation
        ForceCalculationKernel<<<gridDim, blockDim>>>(posx, posy, child, count, sort, deviceEdge, deviceIdx,
                dispx, dispy, node_num, _maxdepth, radius);
        hipDeviceSynchronize();
        thr *= 0.99; //TODO
        //Update
        UpdatePosKernel<<<gridDim, blockDim>>>(posx, posy, dispx, dispy);
        hipDeviceSynchronize();


    }
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(){
    int N = 8, M = 8;
    float *hostx = new float[N];
    float *hosty = new float[N];
    int *E = new int[4*M];
    int *Idx = new int[N]();
	for(int i=0; i<N; ++i){
		Idx[i] = 4*(i+1);
		E[4*i] = i;
		E[4*i] = (i+1)%N;
		E[4*i] = i;
		E[4*i] = (i-1+N)%N;
	}

    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
    for(int i=0; i<N; i++){
        hostx[i] = i*0.1f;//(float(rand())/RAND_MAX-0.5f);
        hosty[i] = i*0.1f;//(float(rand())/RAND_MAX-0.5f);
    }

    BH(hostx, hosty, E, Idx, N, 1);
}
