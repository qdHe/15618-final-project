#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <chrono>
#include <algorithm>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

#define W 10
#define H 10
#define BLOCK_SIZE 256

struct GlobalConstants{
	int N;
	int M;
	float K;

	int Iteration;
	int Thr;
};

class Vertex{
public:
    float x;
    float y;

    Vertex():x((float(rand())/RAND_MAX-0.5)), y((float(rand())/RAND_MAX-0.5)){}
};

class Edge{
public:
    int idx1;
    int idx2;

    Edge():idx1(-1), idx2(-1){}
    Edge(int a, int b):idx1(a), idx2(b){}
};

bool cmp(Edge a, Edge b) {
    if (a.idx1<b.idx1) return true;
    else if (a.idx1 == b.idx1 && a.idx2 < b.idx2) return true;
    return false;
}


__constant__ GlobalConstants deviceParams;
//__device__ __inline__ float max(float a, float b){
//    return (a>b)? a:b;
//}
//__device__ __inline__ float min(float a, float b){
//    return (a<b)? a:b;
//}
__device__ __inline__ float repulsive_force(float dist){
    return deviceParams.K*deviceParams.K/dist/deviceParams.N/100.0;
}

__device__ __inline__ float attractive_force(float dist){
    return dist*dist/deviceParams.K/deviceParams.N;
}

__global__ void kernelForceDirected(float  *V, int *E, int *Idx, float *disp, float thr){


    int v = blockIdx.x*blockDim.x + threadIdx.x;
	if(v >= deviceParams.N) return;
	float disp_x, disp_y;
	float2 end1 = *(float2*)(&V[2*v]), end2;
	//for(int itr=0; itr<deviceParams.Iteration; ++itr){
        //if (itr%(Iteration/10) == 0) cout << "Iteration = " << itr+1 << endl;
    disp_x = 0;
    disp_y = 0;
    end1 = *(float2*)(&V[2*v]);
    for(int u=0; u<deviceParams.N; ++u){
        end2 = *(float2*)(&V[2*u]);
        float d_x = end1.x-end2.x;
        float d_y = end1.y-end2.y;
        float dist = sqrt(d_x*d_x+d_y*d_y);
        dist = max(dist, 0.001);
        float rf = repulsive_force(dist);
        //if(v==10) printf("u=%d, repulsive_force= %f\n", u, rf);
        //if(v%1000 && u%1000) cout<<rf<<' '<<af<<' '<<dist<<endl;
        disp_x += d_x/dist*rf;//disp_x
        disp_y += d_y/dist*rf;//disp_y
    }
    int start = 0;
    if (v > 0){
        start = Idx[v-1];
    }
    for(int e=start; e<Idx[v]; ++e){
        int u = E[2*e+1];
        end2 = *(float2*)(&V[2*u]);
        float d_x = end1.x-end2.x;
        float d_y = end1.y-end2.y;
        float dist = sqrt(d_x*d_x+d_y*d_y);
        dist = max(dist, 0.001);
        float af = attractive_force(dist);
        //if(v==24) printf("e=%d, u=%d, att_force= %f\n", e, u, af);
        disp_x -= d_x/dist*af;
        disp_y -= d_y/dist*af;
    }

    disp[2*v] = disp_x;
    disp[2*v+1] = disp_y;
}

__global__ void kernelUpdate(float* V, float* disp){
    float  thr = deviceParams.Thr;
    int v = threadIdx.x + blockIdx.x * blockDim.x;
    float2 end1 = *(float2*)(&V[2*v]);
    float disp_x = disp[2*v], disp_y = disp[2*v+1];
    float dist = sqrt(disp_x*disp_x + disp_y*disp_y);
    end1.x += (dist > thr)? disp_x/dist*thr : disp_x;
    end1.y += (dist > thr)? disp_y/dist*thr : disp_y;
    end1.x = min(W/2., max(-W/2.,end1.x));
    end1.y = min(H/2., max(-H/2.,end1.y));
    *(float2*)&V[2*v] = end1;

}



int main(int argc, char* argv[]) {

	using namespace std::chrono;
	typedef std::chrono::high_resolution_clock Clock;
	typedef std::chrono::duration<double> dsec;

	int N, M;
    
	ifstream infile;
    infile.open(argv[1]);
	infile >> N >> M;
	Vertex *V = new Vertex[N];
    Edge *E = new Edge[2*M];
    int *Idx = new int[N]();

    Edge e;
    for(int i=0; i<2*M; i+=2){
        infile >> e.idx1 >> e.idx2;
		//cout<<e.idx1<<' '<<e.idx2<<endl;
        E[i] = e;
        swap(e.idx1, e.idx2);
        E[i+1] = e;
    }
    sort(E, E+2*M, cmp);
	float K = sqrt(1.0*W*H/N);
    cout << "Total Edges = " << M << endl;

    for(int i=0; i<2*M; ++i) {
        Idx[E[i].idx1] += 1;
    }

    for(int i=1; i<N; ++i) {
        Idx[i] += Idx[i-1]; //End Index
    }
	//cout<<"]"<<endl;
    cout << "Complete Initialization" << endl;
    int iteration = atoi(argv[2]);
    int thr = W+H;
    //force_directed(V, E, Idx, N, iteration, thr);

    float *deviceVertex;
    int *deviceEdge;
    int *deviceIdx;
    float *deviceDisp;
    hipMalloc(&deviceVertex, sizeof(float)*2*N);
    hipMalloc(&deviceEdge, sizeof(int)*4*M);
    hipMalloc(&deviceIdx, sizeof(int)*N);
    hipMalloc(&deviceDisp, sizeof(float)*2*N);
    hipMemcpy(deviceVertex, V, sizeof(int)*2*N, hipMemcpyHostToDevice);
    hipMemcpy(deviceEdge, E, sizeof(int)*4*M, hipMemcpyHostToDevice);
    hipMemcpy(deviceIdx, Idx, sizeof(int)*N, hipMemcpyHostToDevice);
	
	GlobalConstants params;
	params.N = N;
	params.M = M;
	params.K = K;
	params.Iteration = iteration;
	params.Thr = thr;
	hipMemcpyToSymbol(HIP_SYMBOL(deviceParams), &params, sizeof(GlobalConstants));
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((N+BLOCK_SIZE-1)/BLOCK_SIZE);
    auto calc_start = Clock::now();
    for(int itr=0; itr<iteration; ++itr) {
        kernelForceDirected << < gridDim, blockDim >> > (deviceVertex, deviceEdge,
                deviceIdx, deviceDisp, thr);
        //thr *= 0.99;
        hipDeviceSynchronize();
        kernelUpdate << < gridDim, blockDim >> > (deviceVertex, deviceDisp);
        hipDeviceSynchronize();
    }
    double calc_time = duration_cast<dsec>(Clock::now() - calc_start).count();
    cout << "Time: " << calc_time << endl;

    hipMemcpy(V, deviceVertex, sizeof(float)*2*N,  hipMemcpyDeviceToHost);
    hipFree(deviceVertex);
    hipFree(deviceEdge);
    hipFree(deviceIdx);


    ofstream outfile("Vertex_Pos_pl.txt");
    for (int v=0; v<N; ++v){
        outfile << V[v].x <<' '<<V[v].y<<endl;
    }
    outfile.close(); 
    return 0;
}
